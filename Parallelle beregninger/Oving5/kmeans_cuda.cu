#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Type for points
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int cluster; // cluster this point belongs to
} Point;

// Type for centroids
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int nPoints; // number of points in this cluster
} Centroid;

// Global variables
int nPoints;   // Number of points
int nClusters; // Number of clusters/centroids

Point* points;       // Array containig all points
Centroid* centroids; // Array containing all centroids
Centroid* centroids_temp; //
Centroid* centroids_other; //Array used to reset the centroids, and increment them. 


// Reading command line arguments
void parse_args(int argc, char** argv){
    if(argc != 3){
        printf("Useage: kmeans nClusters nPoints\n");
        exit(-1);
    }
    nClusters = atoi(argv[1]);
    nPoints = atoi(argv[2]);
}


// Create random point
Point create_random_point(){
    Point p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.cluster = rand() % nClusters;
    return p;
}


// Create random centroid
Centroid create_random_centroid(){
    Centroid p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.nPoints = 0;
    return p;
}

Centroid create_random_centroid2(){
    Centroid p;
    p.x = 500.0;
    p.y = 500.0;
    p.nPoints = 0;
    return p;
}




// Initialize random data
// Points will be uniformly distributed
void init_data(){
    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }
    centroids_other = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}

// Initialize random data
// Points will be placed in circular clusters 
void init_clustered_data(){
    float diameter = 500.0/sqrt(nClusters);

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }

    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    for(int i = 0; i < nPoints; i++){
        int c = points[i].cluster;
        points[i].x = centroids[c].x + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].y = centroids[c].y + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].cluster = rand() % nClusters;
    }

    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}


// Print all points and centroids to standard output
void print_data(){
    for(int i = 0; i < nPoints; i++){
        printf("%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    printf("\n\n");
    for(int i = 0; i < nClusters; i++){
        printf("%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }
}

// Print all points and centroids to a file
// File name will be based on input argument
// Can be used to print result after each iteration
void print_data_to_file(int i){
    char filename[15];
    sprintf(filename, "%04d.dat", i);
    FILE* f = fopen(filename, "w+");

    for(int i = 0; i < nPoints; i++){
        fprintf(f, "%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    fprintf(f,"\n\n");
    for(int i = 0; i < nClusters; i++){
        fprintf(f,"%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }

    fclose(f);
}



// Computing distance between point and centroid
float distance(Point a, Centroid b){
    float dx = a.x - b.x;
    float dy = a.y - b.y;

    return sqrt(dx*dx + dy*dy);
}

 
__global__ void set_zero_and_calculate(Point *points, Centroid *centroids,
        Centroid *centroids_other) {
    int t_id = threadIdx.x;
    int local_blockId;  
    
    if (blockIdx.x >= gridDim.x/2) local_blockId = blockIdx.x - (gridDim.x/2);
    else local_blockId = blockIdx.x;


    int global_tid = blockDim.x*local_blockId + t_id;
    
    //Half of the blocks should reset the centroids in centroids_other 
    if (blockIdx.x >= gridDim.x/2) {
        Centroid my_centroid = centroids_other[global_tid];
        my_centroid.x = 0.0;
        my_centroid.y = 0.0;
        my_centroid.nPoints= 0;
        centroids_other[global_tid] = my_centroid; 
    }
    //Other half should calculate the new position for the centroids in the centroids array.
    else {
        Centroid my_centroid = centroids[global_tid];
        if(my_centroid.nPoints == 0){
            my_centroid.x = 500.0;
            my_centroid.y = 500.0;
            my_centroid.nPoints = 0;
        }
        else{
            my_centroid.x /= my_centroid.nPoints;
            my_centroid.y /= my_centroid.nPoints;
        }
        centroids[global_tid] = my_centroid;    
    }

}

__global__ void reassign_points(Point *points, Centroid *centroids, 
        Centroid *centroids_other, int nPoints, int nClusters, int *updated) {

    //Max capacity in shared memory.
    const int MAX_CENT = 4096;
    __shared__ Centroid s_centroids[MAX_CENT]; 
    
    //Get the ids
    int t_id = threadIdx.x;
    int global_tid = blockIdx.x*blockDim.x + t_id;
    
    //The point this thread is going to calculate.
    Point *my_point = &points[global_tid];
    
    //Number of centroids that will be loaded on the current iteration of the while-loop.
    int nof_cent_to_load;
    int nof_cent_loaded = 0;
    
    float bestDistance = DBL_MAX;
    int bestCluster = -1;
    do {
        //Calculate how many centroids that should be transfered to shared memory. 
        if (nClusters > MAX_CENT) {
            nof_cent_to_load = MAX_CENT;
            nClusters -= MAX_CENT;
        }
        else {
            nof_cent_to_load = nClusters;
            nClusters = 0;
        }
        //Load the centroids to shared memory
        double interval = ceil((double)nof_cent_to_load/blockDim.x);
        if (t_id < nof_cent_to_load) {
            for (int i = interval*t_id; i < interval*(t_id+1); i++) {
                int index = nof_cent_loaded + i;
                s_centroids[i] = centroids[index];
            }
        }
        //Make sure every thread is done loading.
        __syncthreads();
        //Calculate the best centroid. 
        for (int i = 0; i < nof_cent_to_load; i++) {
            float dx = my_point->x-s_centroids[i].x;
            float dy = my_point->y-s_centroids[i].y;
            float d = sqrt(dx*dx + dy*dy);
            if (d < bestDistance) {
                bestDistance = d;
                bestCluster = i + nof_cent_loaded; 
            }
        }
        nof_cent_loaded += nof_cent_to_load;
        __syncthreads();
    //Check if all centroids have been processed. If not, load the once that haven't.
    } while (nClusters > 0);
    if (bestCluster != my_point->cluster) {
        *updated = 1; 
    }
    my_point->cluster = bestCluster;
    //Add the location to the best centroid, using the centroid_other array. 
    atomicAdd(&(centroids_other[bestCluster].x), (float)my_point->x);
    atomicAdd(&(centroids_other[bestCluster].y), (float)my_point->y);
    atomicAdd(&(centroids_other[bestCluster].nPoints), 1);
}


int main(int argc, char** argv){
    srand(5);
    parse_args(argc, argv);

    // Create random data, either function can be used.
    //init_clustered_data();
    init_data();
    size_t size_p = sizeof(Point)*nPoints;
    size_t size_c = sizeof(Centroid)*nClusters;

    //Allocate space on the device
    Point *c_points;
    Centroid *c_centroids;
    Centroid *c_centroids_other;
    int *c_updated;

    hipMalloc((void**)&c_points, sizeof(Point)*nPoints);
    hipMalloc((void**)&c_centroids, sizeof(Centroid)*nClusters);
    hipMalloc((void**)&c_centroids_other, sizeof(Centroid)*nClusters);
    hipMalloc((void**)&c_updated, sizeof(int));

    //There should be no more than 1024 threads pr. block. 
    int nofBlocks = nPoints/1024;
    int nofBlocks_clusters;
    if (nClusters > 1024) nofBlocks_clusters = nClusters/1024;
    else nofBlocks_clusters = 1;
    // Iterate until no points are updated
    int updated = 1;

    //Compute new centroids positions on centroids
    for(int i = 0; i < nClusters; i++){
            centroids[i].x = 0.0;
            centroids[i].y = 0.0;
            centroids[i].nPoints= 0;
    }

    for(int i = 0; i < nPoints; i++){
        int c = points[i].cluster;
        centroids[c].x += points[i].x;
        centroids[c].y += points[i].y;
        centroids[c].nPoints++;
    }
    //Only need to copy the points once.
    hipMemcpy(c_points, points, size_p, hipMemcpyHostToDevice);
    
    while(updated){
        updated = 0;

        hipMemcpy(c_centroids, centroids, size_c, hipMemcpyHostToDevice);
        hipMemcpy(c_centroids_other, centroids_other, size_c, hipMemcpyHostToDevice);
        hipMemcpy(c_updated, &updated, sizeof(int), hipMemcpyHostToDevice);
        //Reset the positions to all the centroids in centroids_other
        //Calculate the new positions in centroids
        set_zero_and_calculate<<<2*nofBlocks_clusters, nClusters/nofBlocks_clusters>>>(c_points, c_centroids, c_centroids_other);
        
        

        //reassign points using centroids
        //For each point, add the position to the position of the respective centroid, using centroid_others
        reassign_points<<<nofBlocks, nPoints/nofBlocks>>>(c_points, c_centroids, c_centroids_other, nPoints, nClusters, c_updated);
        hipMemcpy(&updated, c_updated, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(centroids, c_centroids, size_c, hipMemcpyDeviceToHost);
        hipMemcpy(centroids_other, c_centroids_other, size_c, hipMemcpyDeviceToHost); 
        
        
        centroids_temp = centroids;
        centroids = centroids_other;
        centroids_other = centroids_temp;

    }
    hipMemcpy(points, c_points, size_p, hipMemcpyDeviceToHost);
    

    centroids = centroids_other;
    
    //Free memory on the device
    hipFree(c_points);
    hipFree(c_centroids);
    hipFree(c_updated);
    hipFree(c_centroids_other);
    print_data_to_file(1);
}
