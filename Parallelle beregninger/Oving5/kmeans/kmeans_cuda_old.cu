#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>

// Type for points
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int cluster; // cluster this point belongs to
} Point;

// Type for centroids
typedef struct{
    float x;    // x coordinate
    float y;    // y coordinate
    int nPoints; // number of points in this cluster
} Centroid;

// Global variables
int nPoints;   // Number of points
int nClusters; // Number of clusters/centroids

Point* points;       // Array containig all points
Centroid* centroids; // Array containing all centroids


// Reading command line arguments
void parse_args(int argc, char** argv){
    if(argc != 3){
        printf("Useage: kmeans nClusters nPoints\n");
        exit(-1);
    }
    nClusters = atoi(argv[1]);
    nPoints = atoi(argv[2]);
}


// Create random point
Point create_random_point(){
    Point p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.cluster = rand() % nClusters;
    return p;
}


// Create random centroid
Centroid create_random_centroid(){
    Centroid p;
    p.x = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.y = ((float)rand() / (float)RAND_MAX) * 1000.0 - 500.0;
    p.nPoints = 0;
    return p;
}




// Initialize random data
// Points will be uniformly distributed
void init_data(){
    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}

// Initialize random data
// Points will be placed in circular clusters 
void init_clustered_data(){
    float diameter = 500.0/sqrt(nClusters);

    centroids = (Centroid*)malloc(sizeof(Centroid)*nClusters);
    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }

    points = (Point*)malloc(sizeof(Point)*nPoints);
    for(int i = 0; i < nPoints; i++){
        points[i] = create_random_point();
        if(i < nClusters){
            points[i].cluster = i;
        }
    }

    for(int i = 0; i < nPoints; i++){
        int c = points[i].cluster;
        points[i].x = centroids[c].x + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].y = centroids[c].y + ((float)rand() / (float)RAND_MAX) * diameter - (diameter/2);
        points[i].cluster = rand() % nClusters;
    }

    for(int i = 0; i < nClusters; i++){
        centroids[i] = create_random_centroid();
    }
}


// Print all points and centroids to standard output
void print_data(){
    for(int i = 0; i < nPoints; i++){
        printf("%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    printf("\n\n");
    for(int i = 0; i < nClusters; i++){
        printf("%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }
}

// Print all points and centroids to a file
// File name will be based on input argument
// Can be used to print result after each iteration
void print_data_to_file(int i){
    char filename[15];
    sprintf(filename, "%04d.dat", i);
    FILE* f = fopen(filename, "w+");

    for(int i = 0; i < nPoints; i++){
        fprintf(f, "%f\t%f\t%d\t\n", points[i].x, points[i].y, points[i].cluster);
    }
    fprintf(f,"\n\n");
    for(int i = 0; i < nClusters; i++){
        fprintf(f,"%f\t%f\t%d\t\n", centroids[i].x, centroids[i].y, i);
    }

    fclose(f);
}



// Computing distance between point and centroid
float distance(Point a, Centroid b){
    float dx = a.x - b.x;
    float dy = a.y - b.y;

    return sqrt(dx*dx + dy*dy);
}

__global__ void increment_centroids(Point *points, Centroid *centroids,
        int nPoints, int nClusters) {
    int t_id = threadIdx.x;
    int global_tid = blockDim.x*blockIdx.x + t_id;
    Point *my_point = &points[global_tid];
    int c = my_point->cluster;
    atomicAdd(&(centroids[c].x), my_point->x);
    atomicAdd(&(centroids[c].y), my_point->y);
    atomicAdd(&(centroids[c].nPoints), 1);




}

__global__ void reassign_points(Point *points, Centroid *centroids, 
        int nPoints, int nClusters, int *updated) {

    const int MAX_CENT = 4096;
    __shared__ Centroid s_centroids[MAX_CENT]; //Max capacity in shared memory.
    int t_id = threadIdx.x;
    int global_tid = blockIdx.x*blockDim.x + t_id;
    Point *my_point = &points[global_tid];
    int nof_cent_to_load;
    int nof_cent_loaded = 0;
    
    float bestDistance = DBL_MAX;
    int bestCluster = -1;
    do {
        //Load centroids into shared memory
        if (nClusters > MAX_CENT) {
            nof_cent_to_load = MAX_CENT;
            nClusters -= MAX_CENT;
        }
        else nof_cent_to_load = nClusters;

        double interval = ceil((double)nof_cent_to_load/blockDim.x);
        if (t_id < nof_cent_to_load) {
            for (int i = interval*t_id; i < interval*(t_id+1); i++) {
                int index = nof_cent_loaded + i;
                s_centroids[i] = centroids[index];
            }
        }
        __syncthreads();
        for (int i = 0; i < nof_cent_to_load; i++) {
            float dx = my_point->x-s_centroids[i].x;
            float dy = my_point->y-s_centroids[i].y;
            float d = sqrt(dx*dx + dy*dy);
            if (d < bestDistance) {
                bestDistance = d;
                bestCluster = i + nof_cent_loaded; 
            }
        }
        nof_cent_loaded += nof_cent_to_load;
    } while (nClusters > MAX_CENT);
    if (bestCluster != my_point->cluster) {
        *updated = 1; 
    }
    my_point->cluster = bestCluster;

}


int main(int argc, char** argv){
    srand(5);
    parse_args(argc, argv);

    // Create random data, either function can be used.
    //init_clustered_data();
    init_data();
    size_t size_p = sizeof(Point)*nPoints;
    size_t size_c = sizeof(Centroid)*nClusters;

    Point *c_points;
    Centroid *c_centroids;
    int *c_updated;

    hipMalloc((void**)&c_points, sizeof(Point)*nPoints);
    hipMalloc((void**)&c_centroids, sizeof(Centroid)*nClusters);
    hipMalloc((void**)&c_updated, sizeof(int));

    //There should be no more than 1024 threads pr. block. 
    int nofBlocks = nPoints/1024;
    // Iterate until no points are updated
    int updated = 1;
    while(updated){
        updated = 0;

        // Reset centroid positions
        for(int i = 0; i < nClusters; i++){
            centroids[i].x = 0.0;
            centroids[i].y = 0.0;
            centroids[i].nPoints= 0;
        }


        // cudaMemcpy(c_points, points, size_p, cudaMemcpyHostToDevice);
        // cudaMemcpy(c_centroids, centroids, size_c, cudaMemcpyHostToDevice);
        
        // increment_centroids<<<nofBlocks, nPoints/nofBlocks>>>(c_points, c_centroids, nPoints, nClusters);
        // cudaMemcpy(points, c_points, size_p, cudaMemcpyDeviceToHost);
        // cudaMemcpy(centroids, c_centroids, size_c, cudaMemcpyDeviceToHost);
        
        //Compute new centroids positions
        for(int i = 0; i < nPoints; i++){
            int c = points[i].cluster;
            centroids[c].x += points[i].x;
            centroids[c].y += points[i].y;
            centroids[c].nPoints++;
        }

        for(int i = 0; i < nClusters; i++){
            // If a centroid lost all its points, we give it a random position
            // (to avoid dividing by 0)
            if(centroids[i].nPoints == 0){
                centroids[i] = create_random_centroid();
            }
            else{
                centroids[i].x /= centroids[i].nPoints;
                centroids[i].y /= centroids[i].nPoints;
            }
        }

        //reassign points
        hipMemcpy(c_points, points, size_p, hipMemcpyHostToDevice);
        hipMemcpy(c_centroids, centroids, size_c, hipMemcpyHostToDevice);
        hipMemcpy(c_updated, &updated, sizeof(int), hipMemcpyHostToDevice);

        reassign_points<<<nofBlocks, nPoints/nofBlocks>>>(c_points, c_centroids, nPoints, nClusters, c_updated);
        hipMemcpy(points, c_points, size_p, hipMemcpyDeviceToHost);
        hipMemcpy(centroids, c_centroids, size_c, hipMemcpyDeviceToHost);
        hipMemcpy(&updated, c_updated, sizeof(int), hipMemcpyDeviceToHost);
        
    }
    hipFree(c_points);
    hipFree(c_centroids);
    hipFree(c_updated);
    print_data_to_file(1);
}
