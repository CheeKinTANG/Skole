#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//#include "bmp.h"
extern "C" void write_bmp(unsigned char* data, int width, int height);
extern "C" unsigned char* read_bmp(char* filename);
//#include "host_blur.h"
extern "C" void host_blur(unsigned char* inputImage, unsigned char* outputImage, int size);

const int DIM_X = 512;
const int DIM_Y = 512;

void print_properties(){
	int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  printf("Device count: %d\n", deviceCount);

	hipDeviceProp_t p;
	hipSetDevice(0);
	hipGetDeviceProperties (&p, 0);
	printf("Compute capability: %d.%d\n", p.major, p.minor);
	printf("Name: %s\n" , p.name);
	printf("\n\n");
}

__global__ void device_blur(unsigned char *A, unsigned char *B) {
    int t_id = threadIdx.x;
    int b_id = blockIdx.x;

    //Shared array containing the pixels in this row, and the ones below and above. 
    __shared__ unsigned char As[3][DIM_X]; 
     

    //Make all the threads load values into the shared memory.
    //Except if you're on the border.  
    if (b_id != 0) As[0][t_id] = A[(b_id-1)*DIM_X + t_id];
    if (b_id != DIM_Y-1) As[2][t_id] = A[(b_id+1)*DIM_X + t_id];
    As[1][t_id] = A[(b_id)*DIM_X + t_id]; 
    //Wait until all the threads are done.
    __syncthreads();

    //Calculate pixel
    unsigned char new_pixel;
    if ((b_id != 0 && b_id != DIM_Y-1) && (t_id != 0 && t_id != DIM_X-1)){
        new_pixel = 0;
        for (int i = 2; i >= 0; i--) {
            for (int j = -1; j < 2; j++) {
                new_pixel += (As[i][t_id+j])/9.0;
            }
        }
    }
    else new_pixel = As[1][t_id];
    //Store in global. 
    B[b_id*DIM_X + t_id] = new_pixel; 
    
}


int main(int argc,char **argv) {
	
    // Prints some device properties, also to make sure the GPU works etc.
    print_properties();

    unsigned char* picture = read_bmp("peppers.bmp");
    unsigned char *A, *B, *C;

    size_t size = sizeof(unsigned char)*DIM_X*DIM_Y;
    //Currently we do the bluring on the CPU
    //host_blur(A, B, 512);
	
    // You need to:

    // 1. Allocate buffers for the input image and the output image
    hipMalloc((void**)&B, size);
    hipMalloc((void**)&A, size);
    // 2. Transfer the input image from the host to the device
    hipMemcpy(A, picture, size, hipMemcpyHostToDevice);
    // 3. Launch the kernel which does the bluring
	device_blur<<<DIM_Y, DIM_X>>>(A, B);
    C = (unsigned char*)malloc(size);
    // 4. Transfer the result back to the host.
    hipMemcpy(C, B, size, hipMemcpyDeviceToHost);
    write_bmp(C, 512, 512);
    hipFree(A);
    hipFree(B);
    free(picture);
    free(C);


	return 0;
}
